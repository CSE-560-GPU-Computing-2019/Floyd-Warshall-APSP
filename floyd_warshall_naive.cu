#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void fw(int *input, int k, int n)
{
	int i = threadIdx.x + blockIdx.x * blockDimx.x;
	int j = threadIdx.y + blockIdx.y * blockDimx.y;

	if(i<n && j<n){
		if(input[i*n+j]==-1)
			input[i*n+j]=input[i*n+k]+input[k*n+j];
		else
			input[i*n+j]=min(input[i*n+k]+input[k*n+j],input[i*n+j]);
	}
}